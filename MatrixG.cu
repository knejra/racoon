#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipfft/hipfft.h"

#define DEFAULT_THREAD_NUM 64
#define GET_BLOCK_NUM(n) (((n) + DEFAULT_THREAD_NUM - 1) / DEFAULT_THREAD_NUM)

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

template<typename Dtype>
__global__ void scalarAddKern(Dtype *dst, const Dtype scalar, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for(; i < n; i++)
    {
        dst[i] += scalar;
    }
}

void scalarAdd(float *dst, const float scalar, int n)
{
    scalarAddKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(dst, scalar, n);
}

void scalarAdd(double *dst, const double scalar, int n)
{
    scalarAddKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(dst, scalar, n);
}

void scalarAddHost(float *dst, const float scalar, int n)
{
    float *devs;
    hipMalloc((void **)&devs, n * sizeof(float));
    hipMemcpy(devs, dst, n * sizeof(float), hipMemcpyHostToDevice);
    scalarAdd(devs, scalar, n);
    hipMemcpy(dst, devs, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devs);
}

void scalarAddHost(double *dst, const double scalar, int n)
{
    double *devs;
    hipMalloc((void **)&devs, n * sizeof(double));
    hipMemcpy(devs, dst, n * sizeof(double), hipMemcpyHostToDevice);
    scalarAdd(devs, scalar, n);
    hipMemcpy(dst, devs, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devs);
}

template<typename Dtype>
__global__ void scalarSubKern(Dtype *dst, const Dtype scalar, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for(; i < n; i++)
    {
        dst[i] -= scalar;
    }
}

void scalarSub(float *dst, const float scalar, int n)
{
    scalarSubKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(dst, scalar, n);
}

void scalarSub(double *dst, const double scalar, int n)
{
    scalarSubKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(dst, scalar, n);
}

void scalarSubHost(float *dst, const float scalar, int n)
{
    float *devs;
    hipMalloc((void **)&devs, n * sizeof(float));
    hipMemcpy(devs, dst, n * sizeof(float), hipMemcpyHostToDevice);
    scalarSub(devs, scalar, n);
    hipMemcpy(dst, devs, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devs);
}

void scalarSubHost(double *dst, const double scalar, int n)
{
    double *devs;
    hipMalloc((void **)&devs, n * sizeof(double));
    hipMemcpy(devs, dst, n * sizeof(double), hipMemcpyHostToDevice);
    scalarSub(devs, scalar, n);
    hipMemcpy(dst, devs, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devs);
}

template<typename Dtype>
__global__ void matrixAddKern(Dtype *A, Dtype *B, Dtype *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i++)
    {
        C[i] = A[i] + B[i];
    }
}

void matrixAdd(float *A, float *B, float *C, int n)
{
    matrixAddKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void matrixAdd(double *A, double *B, double *C, int n)
{
    matrixAddKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void matrixAddHost(float *A, float *B, float *C, int n)
{
    float *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(float));
    hipMalloc((void **)&devB, n * sizeof(float));
    hipMalloc((void **)&devC, n * sizeof(float));
    hipMemcpy(devA, A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(float), hipMemcpyHostToDevice);
    matrixAdd(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void matrixAddHost(double *A, double *B, double *C, int n)
{
    double *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(double));
    hipMalloc((void **)&devB, n * sizeof(double));
    hipMalloc((void **)&devC, n * sizeof(double));
    hipMemcpy(devA, A, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(double), hipMemcpyHostToDevice);
    matrixAdd(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

template<typename Dtype>
__global__ void matrixSubKern(Dtype *A, Dtype *B, Dtype *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i++)
    {
        C[i] = A[i] - B[i];
    }
}

void matrixSub(float *A, float *B, float *C, int n)
{
    matrixSubKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void matrixSub(double *A, double *B, double *C, int n)
{
    matrixSubKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void matrixSubHost(float *A, float *B, float *C, int n)
{
    float *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(float));
    hipMalloc((void **)&devB, n * sizeof(float));
    hipMalloc((void **)&devC, n * sizeof(float));
    hipMemcpy(devA, A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(float), hipMemcpyHostToDevice);
    matrixSub(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void matrixSubHost(double *A, double *B, double *C, int n)
{
    double *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(double));
    hipMalloc((void **)&devB, n * sizeof(double));
    hipMalloc((void **)&devC, n * sizeof(double));
    hipMemcpy(devA, A, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(double), hipMemcpyHostToDevice);
    matrixSub(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

template<typename Dtype>
__global__ void vectorDotKern(Dtype *A, Dtype *B, Dtype *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i++)
    {
        C[i] = A[i] * B[i];
    }
}

void vectorDot(float *A, float *B, float *C, int n)
{
    vectorDotKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void vectorDot(double *A, double *B, double *C, int n)
{
    vectorDotKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void vectorDotHost(float *A, float *B, float *C, int n)
{
    float *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(float));
    hipMalloc((void **)&devB, n * sizeof(float));
    hipMalloc((void **)&devC, n * sizeof(float));
    hipMemcpy(devA, A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(float), hipMemcpyHostToDevice);
    vectorDot(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void vectorDotHost(double *A, double *B, double *C, int n)
{
    double *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(double));
    hipMalloc((void **)&devB, n * sizeof(double));
    hipMalloc((void **)&devC, n * sizeof(double));
    hipMemcpy(devA, A, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(double), hipMemcpyHostToDevice);
    vectorDot(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

template<typename Dtype>
__global__ void vectorDivKern(Dtype *A, Dtype *B, Dtype *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i++)
    {
        C[i] = A[i] / B[i];
    }
}

void vectorDiv(float *A, float *B, float *C, int n)
{
    vectorDivKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void vectorDiv(double *A, double *B, double *C, int n)
{
    vectorDivKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void vectorDivHost(float *A, float *B, float *C, int n)
{
    float *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(float));
    hipMalloc((void **)&devB, n * sizeof(float));
    hipMalloc((void **)&devC, n * sizeof(float));
    hipMemcpy(devA, A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(float), hipMemcpyHostToDevice);
    vectorDiv(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void vectorDivHost(double *A, double *B, double *C, int n)
{
    double *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(double));
    hipMalloc((void **)&devB, n * sizeof(double));
    hipMalloc((void **)&devC, n * sizeof(double));
    hipMemcpy(devA, A, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(double), hipMemcpyHostToDevice);
    vectorDiv(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void matrixGemmHost(float *A, float *B, float *C, int m, int n, int k)
{
    float *devA, *devB, *devC;
    float alpha = 1.0, beta = 0.0;
    hipblasHandle_t handle;
    int status = hipblasCreate(&handle);
    int asize = m * k;
    int bsize = k * n;
    int csize = m * n;
    hipMalloc((void **)&devA, asize * sizeof(float));
    hipMalloc((void **)&devB, bsize * sizeof(float));
    hipMalloc((void **)&devC, csize * sizeof(float));
    status = hipblasSetVector(asize, sizeof(float), A, 1, devA, 1);
    status = hipblasSetVector(bsize, sizeof(float), B, 1, devB, 1);
    status = hipblasSetVector(csize, sizeof(float), C, 1, devC, 1);
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, 
                         &alpha, devB, n, devA, k, &beta, devC, n);
    status = hipblasGetVector(csize, sizeof(float), devC, 1, C, 1);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    status = hipblasDestroy(handle);
}

void matrixGemmHost(double *A, double *B, double *C, int m, int n, int k)
{
    double *devA, *devB, *devC;
    double alpha = 1.0, beta = 0.0;
    hipblasHandle_t handle;
    int status = hipblasCreate(&handle);
    int asize = m * k;
    int bsize = k * n;
    int csize = m * n;
    hipMalloc((void **)&devA, asize * sizeof(double));
    hipMalloc((void **)&devB, bsize * sizeof(double));
    hipMalloc((void **)&devC, csize * sizeof(double));
    status = hipblasSetVector(asize, sizeof(double), A, 1, devA, 1);
    status = hipblasSetVector(bsize, sizeof(double), B, 1, devB, 1);
    status = hipblasSetVector(csize, sizeof(double), C, 1, devC, 1);
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, 
                         &alpha, devB, n, devA, k, &beta, devC, n);
    status = hipblasGetVector(csize, sizeof(double), devC, 1, C, 1);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    status = hipblasDestroy(handle);
}

// float, C = A_T * B
void matrixTransGemmHost(float *A, float *B, float *C, int m, int n, int k)
{
    float *devA, *devB, *devC;
    float alpha = 1.0, beta = 0.0;
    hipblasHandle_t handle;
    int status = hipblasCreate(&handle);
    int asize = m * k;
    int bsize = k * n;
    int csize = m * n;
    hipMalloc((void **)&devA, asize * sizeof(float));
    hipMalloc((void **)&devB, bsize * sizeof(float));
    hipMalloc((void **)&devC, csize * sizeof(float));
    status = hipblasSetVector(asize, sizeof(float), A, 1, devA, 1);
    status = hipblasSetVector(bsize, sizeof(float), B, 1, devB, 1);
    status = hipblasSetVector(csize, sizeof(float), C, 1, devC, 1);
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, m, k, 
                         &alpha, devB, n, devA, m, &beta, devC, n);
    status = hipblasGetVector(csize, sizeof(float), devC, 1, C, 1);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    status = hipblasDestroy(handle);
}

// double, C = A_T * B
void matrixTransGemmHost(double *A, double *B, double *C, int m, int n, int k)
{
    double *devA, *devB, *devC;
    double alpha = 1.0, beta = 0.0;
    hipblasHandle_t handle;
    int status = hipblasCreate(&handle);
    int asize = m * k;
    int bsize = k * n;
    int csize = m * n;
    hipMalloc((void **)&devA, asize * sizeof(double));
    hipMalloc((void **)&devB, bsize * sizeof(double));
    hipMalloc((void **)&devC, csize * sizeof(double));
    status = hipblasSetVector(asize, sizeof(double), A, 1, devA, 1);
    status = hipblasSetVector(bsize, sizeof(double), B, 1, devB, 1);
    status = hipblasSetVector(csize, sizeof(double), C, 1, devC, 1);
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, m, k, 
                         &alpha, devB, n, devA, m, &beta, devC, n);
    status = hipblasGetVector(csize, sizeof(double), devC, 1, C, 1);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    status = hipblasDestroy(handle);
}

// conv_im2col
template <typename Dtype>
__global__ void im2colKern(const int n, const Dtype *pic, const int height, const int width, const int ksize, 
                           const int pad, const int stride, const int colHeight, const int colWidth, Dtype* colPic) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i += blockDim.x * gridDim.x)
    {
        int outWidth = i % colWidth;
        i = i / colWidth;
        int outHeight = i % colHeight;
        int inChan = i / colHeight;
        int outChan = inChan * ksize * ksize;
        int inHeight = outHeight * stride - pad;
        int inWidth = outWidth * stride - pad;
        colPic += (outChan * colHeight + outHeight) * colWidth + outWidth;
        pic += (inChan * height + inHeight) * width + inWidth;
        for (int p = 0; p < ksize; p++) 
        {
            for (int q = 0; q < ksize; q++) 
            {
                *colPic = (inHeight + p >= 0 && inWidth + q >= 0 && inHeight + p < height && inWidth + q < width) ? 
                          pic[p * width + q] : 0;
                colPic += colHeight * colWidth;
            }
        }
    }
}

template <typename Dtype>
__global__ void col2imKern(const int n, const Dtype* colPic, const int height, const int width, 
                           const int channels, const int ksize, const int pad, const int stride, 
                           const int colHeight, const int colWidth, Dtype* pic) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i++)
    {
        Dtype val = 0;
        int w = i % width + pad;
        int h = (i / width) % height + pad;
        int c = i / (width * height);
        int colStartW = (w < ksize) ? 0 : (w - ksize) / stride + 1;
        int colEndW = min(w / stride + 1, colWidth);
        int colStartH = (h < ksize) ? 0 : (h - ksize) / stride + 1;
        int colEndH = min(h / stride + 1, colHeight);
        for (int p = colStartH; p < colEndH; p++) 
        {
            for (int q = colStartW; q < colEndW; q++) 
            {
                int tmp = c * ksize * ksize + (h - p * stride) * ksize + (w - q * stride);
                val = colPic[(tmp * colHeight + p) * colWidth + q];
            }
        }
        pic[i] = val;
  }
}

void im2colConvHost(float *pic, const int height, const int width, float *kern, const int ksize, 
                    const int channels, const int pad, const int stride)
{
    float *devP, *devC, *devK, *devKC;

    int size = height * width * sizeof(float);
    hipMalloc((void **)&devP, size);
    hipMemcpy(devP, pic, size, hipMemcpyHostToDevice);
    int kksize = ksize * ksize * sizeof(float);
    hipMalloc((void **)&devK, kksize);
    hipMemcpy(devK, kern, kksize, hipMemcpyHostToDevice);

    // image to colImage
    int colHeight = (height + 2 * pad - ksize) / stride + 1;
    int colWidth = (width + 2 * pad - ksize) / stride + 1;
    int csize = colHeight * colWidth * channels * ksize * ksize * sizeof(float);
    hipMalloc((void **)&devC, csize);
    int nkern = channels * colHeight * colWidth;
    im2colKern<float><<<GET_BLOCK_NUM(nkern), DEFAULT_THREAD_NUM>>>(nkern, devP, height, width, ksize, pad, stride, 
                                                                    colHeight, colWidth, devC);
    // test
    float *colPic = new float[csize];
    hipMemcpy(colPic, devC, csize, hipMemcpyDeviceToHost);
    for(int i = 0; i < channels * ksize * ksize; i++)
    {
        for(int j = 0; j < colWidth * colHeight; j++)
        {
            printf("%.2lf ", colPic[i * colWidth * colHeight + j]);
        }
        printf("\n");
    }

    // kernel to colKernel
    int colKernSize = (ksize + 2 * pad - ksize) / stride + 1;
    int cksize = colKernSize * colKernSize * channels * ksize * ksize * sizeof(float);
    hipMalloc((void **)&devKC, cksize);
    nkern = channels * colKernSize * colKernSize;
    im2colKern<float><<<GET_BLOCK_NUM(nkern), DEFAULT_THREAD_NUM>>>(nkern, devK, ksize, ksize, ksize, 0, 1, 
                                                                    colKernSize, colKernSize, devKC);
    // test
    float * colKern = new float(cksize);
    hipMemcpy(colKern, devKC, cksize, hipMemcpyDeviceToHost);
    printf("\n");
    for(int i = 0; i < channels * ksize * ksize; i++)
    {
        for(int j = 0; j < colKernSize * colKernSize; j++)
        {
            printf("%.2lf ", colKern[i * colKernSize * colKernSize + j]);
        }
        printf("\n");
    }

    // GEMM
    float alpha = 1.0, beta = 0.0;
    hipblasHandle_t handle;
    int status = hipblasCreate(&handle);
    int k = channels * ksize * ksize;
    int n = colWidth * colWidth;
    int m = colKernSize * colKernSize;
    float *mres, *res;
    hipMalloc((void **)&mres, n * k * sizeof(float));
    hipMalloc((void **)&res, n * k * sizeof(float));
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, m, k, 
                         &alpha, devC, n, devKC, m, &beta, mres, n);
    res = new float[n * k * sizeof(float)];
    hipMemcpy(res, mres, n * k * sizeof(float), hipMemcpyDeviceToHost);
    printf("\n");
    for(int i = 0; i < m; i++)
    {
        for(int j = 0; j < n; j++)
        {
            printf("%.2lf ", pr[i * n + j]);
        }
        printf("\n");
    }
    status = hipblasDestroy(handle);

    // col2im

    hipFree(devP);
    hipFree(devC);
    hipFree(devK);
    hipFree(devKC);
    hipFree(result);
    hipFree(imRes);
}

void im2colHost(float *pic, const int height, const int width, const int channels, 
                const int ksize, const int pad, const int stride)
{
    float *devP, *devC;

    int size = height * width * sizeof(float);
    hipMalloc((void **)&devP, size);
    hipMemcpy(devP, pic, size, hipMemcpyHostToDevice);

    int colHeight = (height + 2 * pad - ksize) / stride + 1;
    int colWidth = (width + 2 * pad - ksize) / stride + 1;
    int csize = colHeight * colWidth * channels * ksize * ksize * sizeof(float);
    hipMalloc((void **)&devC, csize);

    int nkern = channels * colHeight * colWidth;
    im2colKern<float><<<GET_BLOCK_NUM(nkern), DEFAULT_THREAD_NUM>>>(nkern, devP, height, width, ksize, pad, stride, 
                                                                    colHeight, colWidth, devC);
    float *colPic = new float[csize];
    hipMemcpy(colPic, devC, csize, hipMemcpyDeviceToHost);
    for(int i = 0; i < colHeight * colWidth; i++)
    {
        for(int j = 0; j < channels * ksize * ksize; j++)
        {
            printf("%.2lf ", colPic[i * colHeight * colWidth + j]);
        }
        printf("\n");
    }
    float *imPic = new float[size];
    nkern = channels * height * width;
    col2imKern<float><<<GET_BLOCK_NUM(nkern), DEFAULT_THREAD_NUM>>>(nkern, devC, height, width, channels, ksize, pad, stride, 
                                                                    colHeight, colWidth, devP);
    hipMemcpy(imPic, devP, size, hipMemcpyDeviceToHost);
    printf("\n");
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            printf("%.2lf ", imPic[i * width + j]);
        }
        printf("\n");
    }
    hipFree(devP);
    hipFree(devC);
}

void im2colHost(double *pic, const int height, const int width, const int channels, 
                const int ksize, const int pad, const int stride)
{
    double *devP, *devC;
    
    int size = height * width * sizeof(double);
    hipMalloc((void **)&devP, size);
    hipMemcpy(devP, pic, size, hipMemcpyHostToDevice);
    
    int colHeight = (height + 2 * pad - ksize) / stride + 1;
    int colWidth = (width + 2 * pad - ksize) / stride + 1;
    int csize = colHeight * colWidth * channels * ksize * ksize * sizeof(double);
    hipMalloc((void **)&devC, csize);
    
    int nkern = channels * colHeight * colWidth;
    im2colKern<double><<<GET_BLOCK_NUM(nkern), DEFAULT_THREAD_NUM>>>(nkern, devP, height, width, ksize, pad, stride, 
                                                                    colHeight, colWidth, devC);
    double *colPic = new double[csize];
    hipMemcpy(colPic, devC, csize, hipMemcpyDeviceToHost);
    for(int i = 0; i < colHeight * colWidth; i++)
    {
        for(int j = 0; j < channels * ksize * ksize; j++)
        {
            printf("%.2lf ", colPic[i * colHeight * colWidth + j]);
        }
        printf("\n");
    }
    
    hipFree(devP);
    hipFree(devC);
}


__global__ void paddingKern()
{

}

void padding()
{

}

__global__ void complexVectorDotKern(hipfftComplex *A, hipfftComplex *B, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i++)
    {
        A[i].x = A[i].x * B[i].x - A[i].y * B[i].y;
        A[i].y = A[i].x * B[i].y + A[i].y * B[i].x;
    }
}

void matrixConvFFT(float *padPic, float *padKern, int height, int width)
{        
    hipfftReal *realDevP, *realDevK;
    hipfftComplex *cplDevP, *cplDevK;
    int rsize = width * height * sizeof(hipfftReal);
    int csize = width * (height / 2 + 1) * sizeof(hipfftComplex);
    
    hipMalloc((void**)&realDevP, rsize);
    hipMalloc((void**)&realDevK, rsize);

    hipMalloc((void**)&cplDevP, csize);
    hipMalloc((void**)&cplDevK, csize);

    hipMemset(realDevP, 0, rsize);
    hipMemset(realDevK, 0, rsize);

    hipMemcpy(realDevP, padPic, rsize,	hipMemcpyHostToDevice);
    hipMemcpy(realDevK, padKern, rsize, hipMemcpyHostToDevice);
    
    // picture, kernel, result
    hipfftHandle planP, planK, planR;
    hipfftPlan2d(&planP, height, width, HIPFFT_R2C);
    hipfftPlan2d(&planK, height, width, HIPFFT_R2C);
    hipfftPlan2d(&planR, height, width, HIPFFT_C2R);
    
    hipfftExecR2C(planP, realDevP, cplDevP);
    hipfftExecR2C(planK, realDevK, cplDevK);
    
    complexVectorDotKern<<<GET_BLOCK_NUM(ceil(width * (height / 2 + 1))), DEFAULT_THREAD_NUM>>>(cplDevP, cplDevK, width * (height / 2 + 1));

    hipfftExecC2R(planR, cplDevP, realDevP);
    
    hipfftReal* result = new hipfftReal[width * (height / 2 + 1) * 2];
    hipMemcpy(result, realDevP, rsize, hipMemcpyDeviceToHost);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            printf("%.2lf ", (1.0f / (width * height)) * result[i * width + j]);
        }
        printf("\n");
    }

    hipFree(realDevP);
    hipFree(realDevK);
    hipFree(cplDevP);
    hipFree(cplDevK);

    hipfftDestroy(planP);
    hipfftDestroy(planK);
    hipfftDestroy(planR);
}

void matrixConvIm2col()
{
    // im2col: picture, kernel
    // gemm
    // col2im: result
}

void matrixConvFFTHost()
{
    // padding
    // conv
    // unpadding
}

// conv_winograd

int main()
{
    // float a[] = {1, 1, 1, 1, 1, 1, 1, 1, 1};
    // float b[] = {1, 1, 1, 1, 1, 1, 1, 1, 1};
    // matrixConvFFT(a, b, 3, 3);
    float c[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    float k[] = {1, 3, 5, 7};
    im2colConvHost(c, 3, 3, k, 2, 1, 0, 1);
}



