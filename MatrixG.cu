#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>


#include "hipblas.h"

#define DEFAULT_THREAD_NUM 64
#define GET_BLOCK_NUM(n) (((n) + DEFAULT_THREAD_NUM - 1) / DEFAULT_THREAD_NUM)

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

template<typename Dtype>
__global__ void scalarAddKern(Dtype *dst, const Dtype scalar, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for(; i < n; i++)
    {
        dst[i] += scalar;
    }
}

void scalarAdd(float *dst, const float scalar, int n)
{
    scalarAddKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(dst, scalar, n);
}

void scalarAdd(double *dst, const double scalar, int n)
{
    scalarAddKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(dst, scalar, n);
}

void scalarAddHost(float *dst, const float scalar, int n)
{
    float *devs;
    hipMalloc((void **)&devs, n * sizeof(float));
    hipMemcpy(devs, dst, n * sizeof(float), hipMemcpyHostToDevice);
    scalarAdd(devs, scalar, n);
    hipMemcpy(dst, devs, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devs);
}

void scalarAddHost(double *dst, const double scalar, int n)
{
    double *devs;
    hipMalloc((void **)&devs, n * sizeof(double));
    hipMemcpy(devs, dst, n * sizeof(double), hipMemcpyHostToDevice);
    scalarAdd(devs, scalar, n);
    hipMemcpy(dst, devs, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devs);
}

template<typename Dtype>
__global__ void scalarSubKern(Dtype *dst, const Dtype scalar, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for(; i < n; i++)
    {
        dst[i] -= scalar;
    }
}

void scalarSub(float *dst, const float scalar, int n)
{
    scalarSubKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(dst, scalar, n);
}

void scalarSub(double *dst, const double scalar, int n)
{
    scalarSubKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(dst, scalar, n);
}

void scalarSubHost(float *dst, const float scalar, int n)
{
    float *devs;
    hipMalloc((void **)&devs, n * sizeof(float));
    hipMemcpy(devs, dst, n * sizeof(float), hipMemcpyHostToDevice);
    scalarSub(devs, scalar, n);
    hipMemcpy(dst, devs, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devs);
}

void scalarSubHost(double *dst, const double scalar, int n)
{
    double *devs;
    hipMalloc((void **)&devs, n * sizeof(double));
    hipMemcpy(devs, dst, n * sizeof(double), hipMemcpyHostToDevice);
    scalarSub(devs, scalar, n);
    hipMemcpy(dst, devs, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devs);
}

template<typename Dtype>
__global__ void matrixAddKern(Dtype *A, Dtype *B, Dtype *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i++)
    {
        C[i] = A[i] + B[i];
    }
}

void matrixAdd(float *A, float *B, float *C, int n)
{
    matrixAddKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void matrixAdd(double *A, double *B, double *C, int n)
{
    matrixAddKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void matrixAddHost(float *A, float *B, float *C, int n)
{
    float *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(float));
    hipMalloc((void **)&devB, n * sizeof(float));
    hipMalloc((void **)&devC, n * sizeof(float));
    hipMemcpy(devA, A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(float), hipMemcpyHostToDevice);
    matrixAdd(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void matrixAddHost(double *A, double *B, double *C, int n)
{
    double *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(double));
    hipMalloc((void **)&devB, n * sizeof(double));
    hipMalloc((void **)&devC, n * sizeof(double));
    hipMemcpy(devA, A, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(double), hipMemcpyHostToDevice);
    matrixAdd(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

template<typename Dtype>
__global__ void matrixSubKern(Dtype *A, Dtype *B, Dtype *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i++)
    {
        C[i] = A[i] - B[i];
    }
}

void matrixSub(float *A, float *B, float *C, int n)
{
    matrixSubKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void matrixSub(double *A, double *B, double *C, int n)
{
    matrixSubKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void matrixSubHost(float *A, float *B, float *C, int n)
{
    float *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(float));
    hipMalloc((void **)&devB, n * sizeof(float));
    hipMalloc((void **)&devC, n * sizeof(float));
    hipMemcpy(devA, A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(float), hipMemcpyHostToDevice);
    matrixSub(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void matrixSubHost(double *A, double *B, double *C, int n)
{
    double *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(double));
    hipMalloc((void **)&devB, n * sizeof(double));
    hipMalloc((void **)&devC, n * sizeof(double));
    hipMemcpy(devA, A, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(double), hipMemcpyHostToDevice);
    matrixSub(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

template<typename Dtype>
__global__ void vectorDotKern(Dtype *A, Dtype *B, Dtype *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i++)
    {
        C[i] = A[i] * B[i];
    }
}

void vectorDot(float *A, float *B, float *C, int n)
{
    vectorDotKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void vectorDot(double *A, double *B, double *C, int n)
{
    vectorDotKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void vectorDotHost(float *A, float *B, float *C, int n)
{
    float *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(float));
    hipMalloc((void **)&devB, n * sizeof(float));
    hipMalloc((void **)&devC, n * sizeof(float));
    hipMemcpy(devA, A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(float), hipMemcpyHostToDevice);
    vectorDot(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void vectorDotHost(double *A, double *B, double *C, int n)
{
    double *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(double));
    hipMalloc((void **)&devB, n * sizeof(double));
    hipMalloc((void **)&devC, n * sizeof(double));
    hipMemcpy(devA, A, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(double), hipMemcpyHostToDevice);
    vectorDot(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

template<typename Dtype>
__global__ void vectorDivKern(Dtype *A, Dtype *B, Dtype *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; i < n; i++)
    {
        C[i] = A[i] / B[i];
    }
}

void vectorDiv(float *A, float *B, float *C, int n)
{
    vectorDivKern<float><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void vectorDiv(double *A, double *B, double *C, int n)
{
    vectorDivKern<double><<<GET_BLOCK_NUM(n), DEFAULT_THREAD_NUM>>>(A, B, C, n);
}

void vectorDivHost(float *A, float *B, float *C, int n)
{
    float *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(float));
    hipMalloc((void **)&devB, n * sizeof(float));
    hipMalloc((void **)&devC, n * sizeof(float));
    hipMemcpy(devA, A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(float), hipMemcpyHostToDevice);
    vectorDiv(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void vectorDivHost(double *A, double *B, double *C, int n)
{
    double *devA, *devB, *devC;
    hipMalloc((void **)&devA, n * sizeof(double));
    hipMalloc((void **)&devB, n * sizeof(double));
    hipMalloc((void **)&devC, n * sizeof(double));
    hipMemcpy(devA, A, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, n * sizeof(double), hipMemcpyHostToDevice);
    vectorDiv(devA, devB, devC, n);
    hipMemcpy(C, devC, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void matrixGemmHost(float *A, float *B, float *C, int m, int n, int k)
{
    float *devA, *devB, *devC;
    float alpha = 1.0, beta = 0.0;
    hipblasHandle_t handle;
    int status = hipblasCreate(&handle);
    int asize = m * k;
    int bsize = k * n;
    int csize = m * n;
    hipMalloc((void **)&devA, asize * sizeof(float));
    hipMalloc((void **)&devB, bsize * sizeof(float));
    hipMalloc((void **)&devC, csize * sizeof(float));
    status = hipblasSetVector(asize, sizeof(float), A, 1, devA, 1);
    status = hipblasSetVector(bsize, sizeof(float), B, 1, devB, 1);
    status = hipblasSetVector(csize, sizeof(float), C, 1, devC, 1);
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, 
                         &alpha, devB, n, devA, k, &beta, devC, n);
    status = hipblasGetVector(csize, sizeof(float), devC, 1, C, 1);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    status = hipblasDestroy(handle);
}

void matrixGemmHost(double *A, double *B, double *C, int m, int n, int k)
{
    double *devA, *devB, *devC;
    double alpha = 1.0, beta = 0.0;
    hipblasHandle_t handle;
    int status = hipblasCreate(&handle);
    int asize = m * k;
    int bsize = k * n;
    int csize = m * n;
    hipMalloc((void **)&devA, asize * sizeof(double));
    hipMalloc((void **)&devB, bsize * sizeof(double));
    hipMalloc((void **)&devC, csize * sizeof(double));
    status = hipblasSetVector(asize, sizeof(double), A, 1, devA, 1);
    status = hipblasSetVector(bsize, sizeof(double), B, 1, devB, 1);
    status = hipblasSetVector(csize, sizeof(double), C, 1, devC, 1);
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, 
                         &alpha, devB, n, devA, k, &beta, devC, n);
    status = hipblasGetVector(csize, sizeof(double), devC, 1, C, 1);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    status = hipblasDestroy(handle);
}

// float, C = A_T * B
void matrixTransGemmHost(float *A, float *B, float *C, int m, int n, int k)
{
    float *devA, *devB, *devC;
    float alpha = 1.0, beta = 0.0;
    hipblasHandle_t handle;
    int status = hipblasCreate(&handle);
    int asize = m * k;
    int bsize = k * n;
    int csize = m * n;
    hipMalloc((void **)&devA, asize * sizeof(float));
    hipMalloc((void **)&devB, bsize * sizeof(float));
    hipMalloc((void **)&devC, csize * sizeof(float));
    status = hipblasSetVector(asize, sizeof(float), A, 1, devA, 1);
    status = hipblasSetVector(bsize, sizeof(float), B, 1, devB, 1);
    status = hipblasSetVector(csize, sizeof(float), C, 1, devC, 1);
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, m, k, 
                         &alpha, devB, n, devA, m, &beta, devC, n);
    status = hipblasGetVector(csize, sizeof(float), devC, 1, C, 1);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    status = hipblasDestroy(handle);
}

// double, C = A_T * B
void matrixTransGemmHost(double *A, double *B, double *C, int m, int n, int k)
{
    double *devA, *devB, *devC;
    double alpha = 1.0, beta = 0.0;
    hipblasHandle_t handle;
    int status = hipblasCreate(&handle);
    int asize = m * k;
    int bsize = k * n;
    int csize = m * n;
    hipMalloc((void **)&devA, asize * sizeof(double));
    hipMalloc((void **)&devB, bsize * sizeof(double));
    hipMalloc((void **)&devC, csize * sizeof(double));
    status = hipblasSetVector(asize, sizeof(double), A, 1, devA, 1);
    status = hipblasSetVector(bsize, sizeof(double), B, 1, devB, 1);
    status = hipblasSetVector(csize, sizeof(double), C, 1, devC, 1);
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, m, k, 
                         &alpha, devB, n, devA, m, &beta, devC, n);
    status = hipblasGetVector(csize, sizeof(double), devC, 1, C, 1);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    status = hipblasDestroy(handle);
}

int main()
{
    double a[] = {1.0, 2.0, 3.0, 4.0};
    double b[] = {2.0, 3.0, 4.0, 5.0};
    double c[4] = {0};
    matrixTransGemmHost(a, b, c, 2, 2, 2);
    for(int i = 0; i < 4; i++)
    {
        printf("%.2lf\n", c[i]);
    } 
}

// conv_im2col
// conv_fft



